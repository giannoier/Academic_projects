#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
//#define accuracy  	0.05	
#define accuracy  	0.05 


////////////////////////////////////////////////////////////////////////////////
// Row convolution kernel
////////////////////////////////////////////////////////////////////////////////


__global__ void ConvolutionRowGPU(float *d_Dst,float *d_Src,float *d_Filter,int filterR){
  int x =threadIdx.x;
  int y =threadIdx.y;
  int k;
  
  float sum=0;
  for (k = -filterR; k <= filterR; k++) {
    int d = x + k;
    if (d >= 0 && d < blockDim.x) {
      sum += d_Src[y*blockDim.x+d] * d_Filter[filterR- k];
    }     

    d_Dst[y*blockDim.x+x] = sum;
  }
  
}

////////////////////////////////////////////////////////////////////////////////
// Column convolution kernel
////////////////////////////////////////////////////////////////////////////////

     
__global__ void ConvolutionColGPU(float *d_Dst,float *d_Src,float *d_Filter,int filterR){
  int x =threadIdx.x;
  int y =threadIdx.y;
  
  float sum=0;
  
       for (int k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < blockDim.y) {
          sum += d_Src[d * blockDim.x + x] * d_Filter[filterR - k];
        }   
 
        d_Dst[y * blockDim.x + x] = sum;
      }
}

 
////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;
    
    float
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;
    
   
   

    int imageW;
    int imageH;
    unsigned int i;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    
    struct timespec  tv1, tv2;
    
    printf("Enter filter radius : ");
    scanf("%d", &filter_radius);
  

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;
    
    dim3 blockSize(imageW,imageH);
 

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU	= (float *)malloc(imageW * imageH * sizeof(float));
  

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++)
    {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++)
    {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 16);
    }


    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);
    
    printf ("CPU TIME = %g seconds\n",(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +(double) (tv2.tv_sec - tv1.tv_sec));
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
	//orizw to block ws imageW * imageH
    
    
    //desmeusi mnimis stin GPU
    hipMalloc((void**)&d_Filter,FILTER_LENGTH * sizeof(float));
    hipMalloc((void**)&d_Input,imageW * imageH * sizeof(float));
    hipMalloc((void**)&d_Buffer,imageW * imageH * sizeof(float));
    hipMalloc((void**)&d_OutputGPU,imageW * imageH * sizeof(float));
    
    //elegxos an desmeutike i mnimi stin GPU
    if(d_Filter==NULL||d_Input==NULL||d_Buffer==NULL||d_OutputGPU==NULL){
      printf("couldn't allocate memory in GPU\n");
      return 1;
    }
    
    hipEventRecord(start,0);
    
    hipMemcpy(d_Filter,h_Filter,FILTER_LENGTH * sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_Input,h_Input,imageW * imageH * sizeof(float),hipMemcpyHostToDevice);
  
    hipEventRecord(start,0);
    //kernel launch
    ConvolutionRowGPU<<<1,blockSize>>>(d_Buffer, d_Input, d_Filter, filter_radius); // convolution kata grammes  
    
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
   
    if(error != hipSuccess){
      printf("CUDA Error: %s\n", hipGetErrorString(error));
      
      return 1;
    }
    //kernel launch
    ConvolutionColGPU<<<1,blockSize>>>(d_OutputGPU, d_Buffer, d_Filter, filter_radius); // convolution kata sthles
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    //metafora dedomenwn apo tin GPU
    hipMemcpy(h_OutputGPU,d_OutputGPU,imageW * imageH * sizeof(float),hipMemcpyDeviceToHost);
    
    //elegxos gia sfalmata
    hipDeviceSynchronize();
    error = hipGetLastError();
    
    if(error != hipSuccess){
      printf("CUDA Error: %s\n", hipGetErrorString(error));
      
      return 1;
    }
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU TIME = %f\n",milliseconds/1000);
   
     //elegxos apotelesmatos
    i=0;
    while (i<imageW*imageH){
      
      if(ABS(h_OutputGPU[i]-h_OutputCPU[i])>accuracy){
	    printf("Accuracy Error, at element %d\n GPU result - CPU result =  %f\n Aborting...\n",i,h_OutputGPU[i]-h_OutputCPU[i]);
	    break;    
	}
	i++;
    }
   
    
    

    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_OutputGPU);
    
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);
    hipFree(d_Filter);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();

    return 0;
}
